#include "hip/hip_runtime.h"
#include "geometryfusion_mipmap_cpu1.hpp"
#include "mesh_interleaved1.hpp"
#include "mesh_interleaved_meshcell.hpp"
#include "mesh1.hpp"
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudafilters.hpp>
#include <pmmintrin.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <list>
#include "brickz_cuda.hpp"
#include "brick.hpp"

using namespace cv;
using namespace std;


__global__ void add_brickz_kernel (treeinfo info,
                    sidetype ox, sidetype oy, sidetype oz,
                    sidetype size,
                    volumetype lastleaf0, volumetype lastleaf1,
                    const ParentArray &leafParent,
                    const MarchingCubesIndexed &mc,
                    MeshInterleaved *pmesh
        ){
    sidetype &bl = info.brickLength;
	sidetype &brickSize = info.brickSize;
	const sidetype *leafScale = info.leafScale;
	const sidetype3 *leafPos = info.leafPos;
	const float *distance = info.distance;
	const weighttype *weights = info.weights;
	const colortype3 *color = info.color;
	float minWeight = info.minWeight;
//	unsigned int &degenerate_faces = *info.degenerate_faces;
	float3 offset = info.offset;
	float scale = info.scale;
	MeshInterleaved &mesh = *pmesh;

	sidetype bs = bl*bl;

	float *d = new float[brickSize];
	weighttype *w = new weighttype[brickSize];
	VertexColor *c = new VertexColor[brickSize];
	size_t *indices = new size_t[brickSize*3];
	bool *vertexIsSet = new bool[brickSize*3];
	for(unsigned int i=0;i<brickSize*3;i++) vertexIsSet[i] = false;
	bool *faceIsSet = new bool[brickSize*3];
	for(unsigned int i=0;i<brickSize*3;i++) faceIsSet[i] = false;
	int **tables = new int*[brickSize];
	for(unsigned int i=0;i<brickSize;i++) tables[i] = NULL;

	weighttype wf[4]; float df[4]; colortype3 cf[4];

	volumetype lastleaves[2] = {lastleaf0,lastleaf1};
    sidetype sizeMin;
    if(leafScale[lastleaf0]>leafScale[lastleaf1]){
        sizeMin=leafScale[lastleaf1];
    }
    else{
        sizeMin=leafScale[lastleaf0];
    }
	//sidetype sizeMin = std::min(leafScale[lastleaf0],leafScale[lastleaf1]);
	sidetype sizeStretch = leafScale[lastleaf0];

	if(sizeStretch>size){
    //fprintf(stderr,"\nWARNING Wall Z: The Leaf is too large: %i > %i",sizeStretch,size);
		return;
	}

	sidetype z[2] = {(sidetype)(oz+size-sizeStretch) , (sidetype)(oz+size)};
	
	int x1 = threadIdx.x + blockIdx.x*blockDim.x;
	int y1 = threadIdx.y + blockIdx.y*blockDim.y;
	sidetype x=ox+x1*size,y=oy+y1*size;

	if (y<oy+size){
		sidetype by = (y-oy)/sizeMin;
		if(x<ox+size){
			sidetype bx = (x-ox)/sizeMin;

			for(unsigned int bz=0;bz<2;bz++){
				volumetype idx = (bz*bl+by)*bl+bx;
				w[idx] = 0;
				for(volumetype leaf=lastleaves[bz];leaf<BRANCHINIT
#ifdef ADD_WEIGHTS_TRANSITION_140424
				&& w[idx]<=MIN_WEIGHT_FOR_SURFACE;
#else
				&& w[idx]<=0.0f;
#endif
				leaf=leafParent[leaf]
	//				leaf=BRANCHINIT
				 ){
					volumetype start = leaf*brickSize;
					sidetype3 lo = leafPos[leaf];
					sidetype ls = leafScale[leaf];

					sidetype lz = (z[bz]-lo.z)/ls;

					sidetype lxl = (x-lo.x)/ls;
					sidetype lxr = (x-lo.x)%ls;
					float rx = (float)lxr/(float)ls;
					sidetype lxh = lxl+(rx>0.0 && lxl<bl-1);

					sidetype lyl = (y-lo.y)/ls;
					sidetype lyr = (y-lo.y)%ls;
					float ry = (float)lyr/(float)ls;
					sidetype lyh = lyl+(ry>0.0 && lyl<bl-1);

					volumetype idxLeaf[4] = {
							(volumetype)((lz*bl+lyl)*bl+lxl),
							(volumetype)((lz*bl+lyl)*bl+lxh),
							(volumetype)((lz*bl+lyh)*bl+lxl),
							(volumetype)((lz*bl+lyh)*bl+lxh)
					};

					for(volumetype i=0;i<4;i++) {
						wf[i] = weights[start+idxLeaf[i]];
						df[i] = distance[start+idxLeaf[i]];
						if(color) cf[i] = color[start+idxLeaf[i]];
					}

					float rxInv = 1.0f-rx;
					float ryInv = 1.0f-ry;

#ifndef WEIGHT_MINIMUM
#ifdef ADD_WEIGHTS_TRANSITION_140424
					w[idx] += ryInv*rxInv*(float)wf[0]+
									  ryInv*rx   *(float)wf[1]+
									  ry   *rxInv*(float)wf[2]+
									  ry   *rx   *(float)wf[3];
#else
					w[idx] = ryInv*rxInv*(float)wf[0]+
									 ryInv*rx   *(float)wf[1]+
									 ry   *rxInv*(float)wf[2]+
									 ry   *rx   *(float)wf[3];
#endif
#else
					w[idx] = std::min(std::min(wf[0],wf[1]),std::min(wf[2],wf[3]));
#endif

					d[idx] = ryInv*rxInv*df[0]+
									 ryInv*rx   *df[1]+
									 ry   *rxInv*df[2]+
									 ry   *rx   *df[3];

					if(color)
#ifndef BRICKVISUALIZATION
#ifndef COLORINVERSION
					c[idx] = VertexColor(
									 ryInv*rxInv*(float)cf[0].x+
									 ryInv*rx   *(float)cf[1].x+
									 ry   *rxInv*(float)cf[2].x+
									 ry   *rx   *(float)cf[3].x,
									 ryInv*rxInv*(float)cf[0].y+
									 ryInv*rx   *(float)cf[1].y+
									 ry   *rxInv*(float)cf[2].y+
									 ry   *rx   *(float)cf[3].y,
									 ryInv*rxInv*(float)cf[0].z+
									 ryInv*rx   *(float)cf[1].z+
									 ry   *rxInv*(float)cf[2].z+
									 ry   *rx   *(float)cf[3].z);
#else
					c[idx] = VertexColor(
									 ryInv*rxInv*(float)cf[0].z+
									 ryInv*rx   *(float)cf[1].z+
									 ry   *rxInv*(float)cf[2].z+
									 ry   *rx   *(float)cf[3].z,
									 ryInv*rxInv*(float)cf[0].y+
									 ryInv*rx   *(float)cf[1].y+
									 ry   *rxInv*(float)cf[2].y+
									 ry   *rx   *(float)cf[3].y,
									 ryInv*rxInv*(float)cf[0].x+
									 ryInv*rx   *(float)cf[1].x+
									 ry   *rxInv*(float)cf[2].x+
									 ry   *rx   *(float)cf[3].x);
#endif
#else
						c[idx] = VertexColor(0,0,65280);
#endif
				}
			}
		}
	}

	if(y<oy+size-sizeMin){
		sidetype by = (y-oy)/sizeMin;
		if(x<ox+size-sizeMin){
			sidetype bx = (x-ox)/sizeMin;

			volumetype idx = by*bl+bx;
			if(weightInfluence(minWeight,
					w[idx],w[idx+1],
					w[idx+bl+1],w[idx+bl],
					w[idx+bs],w[idx+bs+1],
					w[idx+bs+bl+1],w[idx+bs+bl])){
				int *table = mc.offsetTable[mc.getCubeIndex(
						d[idx],d[idx+1],
						d[idx+bl+1],d[idx+bl],
						d[idx+bs],d[idx+bs+1],
						d[idx+bs+bl+1],d[idx+bs+bl],
						w[idx],w[idx+1],
						w[idx+bl+1],w[idx+bl],
						w[idx+bs],w[idx+bs+1],
						w[idx+bs+bl+1],w[idx+bs+bl])];

				for (unsigned int i=0;table[i]!=-1;i+=3) {
					faceIsSet[3*idx+table[i  ]] = true;
					faceIsSet[3*idx+table[i+1]] = true;
					faceIsSet[3*idx+table[i+2]] = true;
				}
				tables[idx] = table;
			}
		}
	}

	size_t runningIndex = mesh.vertices.size();
	if(y<oy+size){
		sidetype by = (y-oy)/sizeMin;
		if(x<ox+size){
			sidetype bx = (x-ox)/sizeMin;
			for(sidetype bz=0;bz<2;bz++){
				volumetype idx = (bz*bl+by)*bl+bx;

				if(
						x<ox+size-sizeMin &&
						faceIsSet[3*idx+0]
				 	&& w[idx] && w[idx+1] && ((d[idx]<0)!=(d[idx+1]<0))
				){
					Vertex3f ver = MarchingCubes::VertexInterp(
							Vertex3f(offset.x+(x        )*scale,offset.y+y*scale,offset.z+z[bz]*scale),
							Vertex3f(offset.x+(x+sizeMin)*scale,offset.y+y*scale,offset.z+z[bz]*scale),
							d[idx],d[idx+1]);
					VertexColor col = MarchingCubes::VertexInterp(c[idx],c[idx+1],d[idx],d[idx+1]);
					mesh.vertices.push_back(ver);
					mesh.colors.push_back(Color3b(col.x/COLOR_MULTIPLICATOR,col.y/COLOR_MULTIPLICATOR,col.z/COLOR_MULTIPLICATOR));
					indices[3*idx+0] = runningIndex++;
					vertexIsSet[3*idx+0] = true;
				}
				if(
						y<oy+size-sizeMin &&
						faceIsSet[3*idx+1]
						&& w[idx] && w[idx+bl] && ((d[idx]<0)!=(d[idx+bl]<0))
				){
					Vertex3f ver = MarchingCubes::VertexInterp(
							Vertex3f(offset.x+x*scale,offset.y+(y        )*scale,offset.z+z[bz]*scale),
							Vertex3f(offset.x+x*scale,offset.y+(y+sizeMin)*scale,offset.z+z[bz]*scale),
							d[idx],d[idx+bl]);
					VertexColor col = MarchingCubes::VertexInterp(c[idx],c[idx+bl],d[idx],d[idx+bl]);
					mesh.vertices.push_back(ver);
					mesh.colors.push_back(Color3b(col.x/COLOR_MULTIPLICATOR,col.y/COLOR_MULTIPLICATOR,col.z/COLOR_MULTIPLICATOR));
					indices[3*idx+1] = runningIndex++;
					vertexIsSet[3*idx+1] = true;
				}
				if(
					 faceIsSet[3*idx+2]
						&& w[idx] && w[idx+bs] &&((d[idx]<0)!=(d[idx+bs]<0))
				){
					Vertex3f ver = MarchingCubes::VertexInterp(
							Vertex3f(offset.x+x*scale,offset.y+y*scale,offset.z+(z[bz]            )*scale),
							Vertex3f(offset.x+x*scale,offset.y+y*scale,offset.z+(z[bz]+sizeStretch)*scale),
							d[idx],d[idx+bs]);
					VertexColor col = MarchingCubes::VertexInterp(c[idx],c[idx+bs],d[idx],d[idx+bs]);
					mesh.vertices.push_back(ver);
					mesh.colors.push_back(Color3b(col.x/COLOR_MULTIPLICATOR,col.y/COLOR_MULTIPLICATOR,col.z/COLOR_MULTIPLICATOR));
					indices[3*idx+2] = runningIndex++;
					vertexIsSet[3*idx+2] = true;
				}
			}
		}
	}

	if(y<oy+size-sizeMin){
		sidetype by = (y-oy)/sizeMin;
		if(x<ox+size-sizeMin){
			sidetype bx = (x-ox)/sizeMin;

			volumetype idx = by*bl+bx;
			int *table = tables[idx];
			if(table){
				for (unsigned int i=0;table[i]!=-1;i+=3) {
					mesh.faces.push_back(indices[3*idx+table[i  ]]);
					mesh.faces.push_back(indices[3*idx+table[i+1]]);
					mesh.faces.push_back(indices[3*idx+table[i+2]]);
				}
			}
		}
	}

	bool loneVertices = false;
	bool wrongIndices = false;
	int lastWrongIndex = -1;
	for(unsigned int i=0;i<brickSize*3;i++) loneVertices |= (vertexIsSet[i]&& !faceIsSet[i]);
	for(unsigned int i=0;i<brickSize*3;i++) {
		wrongIndices |= (!vertexIsSet[i]&& faceIsSet[i]);
		if(!vertexIsSet[i]&& faceIsSet[i]) lastWrongIndex = i;
	}

    //if(loneVertices) fprintf(stderr,"\nERROR: There were lone Vertices at [%i %i %i]",ox,oy,oz);
	//if(wrongIndices) fprintf(stderr,"\nERROR: There were wrong Indices at [%i %i %i]:%i - > [%i %i %i]%i",
			//ox,oy,oz,lastWrongIndex/3,
			//(lastWrongIndex/3)%bl,((lastWrongIndex/3)/bl)%bl,(lastWrongIndex/3)/bs,lastWrongIndex%3);

	delete [] d; delete [] w; delete [] c;
	delete [] indices;
	delete [] vertexIsSet; delete [] faceIsSet;
	delete [] tables;

            
}






void add_brickz_caller(treeinfo info,
                    sidetype ox, sidetype oy, sidetype oz,
                    sidetype size,
                    volumetype lastleaf0, volumetype lastleaf1,
                    const ParentArray &leafParent,
                    const MarchingCubesIndexed &mc,
                    MeshInterleaved *pmesh
        ){
        dim3 block(16, 16);
        const sidetype *leafScale = info.leafScale;
        sidetype sizeMin = std::min(leafScale[lastleaf0],leafScale[lastleaf1]);
        dim3 grid(((size/sizeMin)+ block.x-1)/ block.x , ((size/sizeMin) + block.y-1)/ block.y);
        add_brickz_kernel <<<grid, block>>>(info, ox, oy, oz,size,lastleaf0,lastleaf1, leafParent,mc, pmesh );
} 
